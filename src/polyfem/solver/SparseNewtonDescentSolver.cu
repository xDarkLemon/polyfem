#include "hip/hip_runtime.h"
#pragma once

#include <polyfem/utils/Logger.hpp>
#include <polyfem/utils/Timer.hpp>
#include <polyfem/Common.hpp>
// #include "NonlinearSolver.hpp"
#include <polysolve/LinearSolver.hpp>
#include <polyfem/utils/MatrixUtils.hpp>

#include <polyfem/solver/NLProblem.hpp>
#include <polyfem/solver/ALNLProblem.hpp>
#include <polyfem/solver/SparseNewtonDescentSolver.hpp>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "polyfem/assembler/CUDA_utilities.cuh"
#include "hipblas.h"

namespace cppoptlib
{
    def check_cuda_mem()
    {
        static int flag_gpu_settings = 0;
        if (!flag_gpu_settings)
        {
            size_t free_bytes = 0, total_bytes = 0;
            hipMemGetInfo(&free_bytes, &total_bytes);
            std::cout << "Mem GPU Free : " << free_bytes << " bytes" << std::endl;
            std::cout << "Mem GPU Total: " << total_bytes << " bytes" << std::endl;
            size_t sizeLimit = 0;
            hipDeviceGetLimit(&sizeLimit, hipLimitMallocHeapSize);
            std::cout << "Original device heap sizeLimit: " << sizeLimit << std::endl;

            //hipDeviceSetLimit( hipLimitMallocHeapSize, free_bytes );

            //hipDeviceGetLimit( &sizeLimit, hipLimitMallocHeapSize );
            //std::cout << "Current device heap sizeLimit: " << sizeLimit << std::endl;
            /*
            std::cout << "SharedMemoryRequired: "
            << ":" << size_inner_index*sizeof(double)
            << std::endl;
            */
            flag_gpu_settings++;
        }
    }

	template <typename ProblemType>
	bool SparseNewtonDescentSolver<ProblemType>::compute_update_direction_cuda(
			ProblemType &objFunc,
			const Eigen::Matrix<double, -1, 1> &x,
			const Eigen::Matrix<double, -1, 1> &grad,
			Eigen::Matrix<double, -1, 1> &direction)
	{
        if (this->descent_strategy == 2)
        {
            direction = -grad;
            return true;
        }

        {
            POLYFEM_SCOPED_TIMER("assembly time", this->assembly_time);

            if (this->descent_strategy == 1)
                objFunc.set_project_to_psd(true);
            else if (this->descent_strategy == 0)
                objFunc.set_project_to_psd(false);
            else
                assert(false);

            objFunc.hessian(x, hessian);

            if (reg_weight > 0)
            {
                hessian += reg_weight * polyfem::utils::sparse_identity(hessian.rows(), hessian.cols());
            }
        }

        {
            POLYFEM_SCOPED_TIMER("linear solve", this->inverting_time);
            // TODO: get the correct size
            linear_solver->analyzePattern(hessian, hessian.rows());

            try
            {
                linear_solver->factorize(hessian);
            }
            catch (const std::runtime_error &err)
            {
                increase_descent_strategy();
                // warn if using gradient descent
                polyfem::logger().log(
                    this->descent_strategy == 2 ? spdlog::level::warn : spdlog::level::debug,
                    "Unable to factorize Hessian: \"{}\"; reverting to {}",
                    err.what(), this->descent_strategy_name());
                // polyfem::write_sparse_matrix_csv("problematic_hessian.csv", hessian);
                return compute_update_direction_cuda(objFunc, x, grad, direction);
            }

            linear_solver->solve(-grad, direction); // H Δx = -g
        }

        // gradient descent, check descent direction
        int N = hessian.cols();
        
        double *hessian_dev, *direction_dev, *grad_dev, *tmp_dev, *res_dev;  // to compute residual
        double *grad_grad_dev, *grad_direc_dev;  // to compute grad norm and grad dot direction
        
        double *hessian_host = hessian.valuePtr();
        double *direction_host = direction.data();
        const double *grad_host = grad.data();
        double *tmp_host = new double[N];
        double *res_host = new double[1];
        double *grad_grad_host = new double[1];
        double *grad_direc_host = new double[1];
        
        double alpha = 1.0;
        double beta = 0.0;
        
        printf("before allocating hessian_dev:\n");
        check_cuda_mem();
        hessian_dev = ALLOCATE_GPU<double>(hessian_dev, N*N*sizeof(double));
        printf("after allocating hessian_dev:\n");
        check_cuda_mem();

        direction_dev = ALLOCATE_GPU<double>(direction_dev, N*sizeof(double));
        grad_dev = ALLOCATE_GPU<double>(grad_dev, N*sizeof(double));
        tmp_dev = ALLOCATE_GPU<double>(tmp_dev, N*sizeof(double));
        res_dev = ALLOCATE_GPU<double>(res_dev, sizeof(double));
        grad_grad_dev = ALLOCATE_GPU<double>(grad_grad_dev, sizeof(double));
        grad_direc_dev = ALLOCATE_GPU<double>(grad_grad_dev, sizeof(double));

        COPYDATATOGPU<double>(hessian_dev, hessian_host, N*N*sizeof(double));
        COPYDATATOGPU<double>(direction_dev, direction_host, N*sizeof(double));
        COPYDATATOGPU<double>(grad_dev, grad_host, N*sizeof(double));
        
        hipblasHandle_t handle;
        hipblasCreate(&handle);
        
        hipDeviceSynchronize();
        hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 1, N, N, &alpha, direction_dev, 1, hessian_dev, N, &beta, tmp_dev, 1);  // hessian * direction
        hipDeviceSynchronize();
        hipblasDaxpy(handle, N, &alpha, grad_dev, 1, tmp_dev, 1); // vector add, hessian * direction + grad
        hipDeviceSynchronize();
        hipblasDdot(handle, N, tmp_dev, 1, tmp_dev, 1, res_dev);  // dot product
        hipDeviceSynchronize();
        
        COPYDATATOHOST<double>(res_host, res_dev, sizeof(double));
        const double residual = std::sqrt(*res_host);  // norm

        hipblasDdot(handle, N, grad_dev, 1, grad_dev, 1, grad_grad_dev);
        hipblasDdot(handle, N, grad_dev, 1, direction_dev, 1, grad_direc_dev);
        COPYDATATOHOST<double>(grad_grad_host, grad_grad_dev, sizeof(double));
        COPYDATATOHOST<double>(grad_direc_host, grad_direc_dev, sizeof(double));
        const double grad_norm = std::sqrt(*grad_grad_host);
        const double grad_direc_prod = *grad_direc_host;
        
        delete[] res_host;
        delete[] tmp_host;
        delete[] grad_grad_host;
        delete[] grad_direc_host;
        hipFree(hessian_dev);
        hipFree(direction_dev);
        hipFree(grad_dev);
        hipFree(tmp_dev);
        hipFree(res_dev);
        hipblasDestroy(handle);

        // gradient descent, check descent direction
        // const double residual = (hessian * direction + grad).norm(); // H Δx + g = 0

        if (std::isnan(residual))
        {
            increase_descent_strategy();
            polyfem::logger().log(
                this->descent_strategy == 2 ? spdlog::level::warn : spdlog::level::debug,
                "nan linear solve residual {} (||∇f||={}); reverting to {}",
                residual, grad_norm, this->descent_strategy_name());
            return compute_update_direction_cuda(objFunc, x, grad, direction);
        }
        else if (residual > std::max(1e-8 * grad_norm, 1e-5))
        {
            increase_descent_strategy();
            polyfem::logger().log(
                this->descent_strategy == 2 ? spdlog::level::warn : spdlog::level::debug,
                "large linear solve residual {} (||∇f||={}); reverting to {}",
                residual, grad_norm, this->descent_strategy_name());
            return compute_update_direction_cuda(objFunc, x, grad, direction);
        }
        else
        {
            polyfem::logger().trace("linear solve residual {}", residual);
        }

        // do this check here because we need to repeat the solve without resetting reg_weight
        if (grad_direc_prod >= 0)
        {
            increase_descent_strategy();
            polyfem::logger().log(
                this->descent_strategy == 2 ? spdlog::level::warn : spdlog::level::debug,
                "[{}] direction is not a descent direction (Δx⋅g={}≥0); reverting to {}",
                name(), direction.dot(grad), descent_strategy_name());
            return compute_update_direction_cuda(objFunc, x, grad, direction);
        }

        json info;
        linear_solver->getInfo(info);
        internal_solver_info.push_back(info);

        reg_weight /= reg_weight_dec;
        if (reg_weight < reg_weight_min)
            reg_weight = 0;

        return true;
    }
    template class SparseNewtonDescentSolver<polyfem::solver::NLProblem>;
    template class SparseNewtonDescentSolver<polyfem::solver::ALNLProblem>;
}