#include "hip/hip_runtime.h"
#include <polyfem/NeoHookeanElasticity.hpp>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#define NUMBER_THREADS  32

#include <polyfem/Basis.hpp>
#include <polyfem/auto_elasticity_rhs.hpp>

#include <polyfem/MatrixUtils.hpp>

namespace polyfem
{

//	__global__ void set_dispv(const Local2Global **bvs_data, Eigen::Matrix<size_t, Eigen::Dynamic,1> &bvs_sizes , const Eigen::MatrixXd &displacement , int size,int bvs_total_size, double* local_disp)
	__global__ void set_dispv(Eigen::Matrix<const Local2Global*, Eigen::Dynamic,1> &bvs_data, Eigen::Matrix<size_t, Eigen::Dynamic,1> &bvs_sizes , const Eigen::MatrixXd &displacement , int size,int bvs_total_size, double* local_disp)
	{
   		int bx = blockIdx.x;
   		int tx = threadIdx.x; 
		int inner_index = bx * NUMBER_THREADS + tx;

		if(inner_index < bvs_total_size)
		{
			for (size_t ii = 0; ii < bvs_sizes(inner_index,1); ++ii)
			{
				for (int d = 0; d < size; ++d)
				{
					local_disp[inner_index * size + d] += bvs_data(ii,1)->val * displacement(bvs_data(ii,1)->index * size + d);
				}
			}
		}

//race_condition
/*		for (int d = 0; d < size; ++d)
		{
			local_disp[inner_index*size + d] += bs_global_val * displacement(bvs_global_index * size + d);
		}
*/
		return;
	}

	double NeoHookeanElasticity::compute_energy(const ElementAssemblyValues &vals, const Eigen::MatrixXd &displacement, const QuadratureVector &da) const 
	{
		return compute_energy_aux<double>(vals, displacement, da);
	}

	// Compute ∫ ½μ (tr(FᵀF) - 3 - 2ln(J)) + ½λ ln²(J) du
	template <typename T>
	T NeoHookeanElasticity::compute_energy_aux(const ElementAssemblyValues &vals, const Eigen::MatrixXd &displacement, const QuadratureVector &da) const
	{
		typedef Eigen::Matrix<T, Eigen::Dynamic, 1> AutoDiffVect;
		typedef Eigen::Matrix<T, Eigen::Dynamic, Eigen::Dynamic, 0, 3, 3> AutoDiffGradMat;

//	    printf("Starting..\n");
		assert(displacement.cols() == 1);

		double *local_dispv_dev=NULL;

		size_t basisvalues_size= vals.basis_values.size();
		size_t data_size= basisvalues_size * size() * sizeof(double); 

		double *local_dispv = new double[vals.basis_values.size()*size()];

		if (hipMalloc((void **)&local_dispv_dev , data_size) != hipSuccess)
 		{
	      printf("Error allocating to GPU\n");
	      abort();
		}
		hipDeviceSynchronize();
		const int n_pts = da.size();

//		Eigen::Matrix<double, Eigen::Dynamic, 1> local_dispv(vals.basis_values.size() * size(), 1);
//		local_dispv.setZero();

		const Local2Global *bs_global;

		const AssemblyValues *bs_storage = vals.basis_values.data();
		//const Local2Global **bs_global_data = NULL;

		Eigen::Matrix<const Local2Global* , Eigen::Dynamic,1> bs_global_data(basisvalues_size,1);
		Eigen::Matrix<size_t, Eigen::Dynamic,1> bs_global_sizes(basisvalues_size,1);
		//int *bs_global_sizes = new int[basisvalues_size];

		for (size_t i = 0; i < basisvalues_size; ++i)
		{
			//bs_global_data[i] = bs_storage[i].global.data();
			bs_global_data(i,1) = bs_storage[i].global.data();
			bs_global_sizes(i,1) = bs_storage[i].global.size();
		}	

		size_t grid_x = (basisvalues_size%NUMBER_THREADS==0) ? basisvalues_size/NUMBER_THREADS : basisvalues_size/NUMBER_THREADS +1;

		set_dispv<<<grid_x,NUMBER_THREADS>>>(bs_global_data, bs_global_sizes, displacement, size(), basisvalues_size, local_dispv_dev);

		hipDeviceSynchronize();
/*
		for (size_t i = 0; i < basisvalues_size; ++i)
		{
			const auto &bs = vals.basis_values[i];
			for (size_t ii = 0; ii < bs.global.size(); ++ii)
			{
				for (int d = 0; d < size(); ++d)
				{
					local_dispv(i * size() + d) += bs.global[ii].val * displacement(bs.global[ii].index * size() + d);
				}
			}

		}
*/
	    if(hipMemcpy(local_dispv,local_dispv_dev,data_size,hipMemcpyDeviceToHost) != hipSuccess)
	    {
		      printf("Error copying to CPU\n");
		      abort(); 
	    }
		

		DiffScalarBase::setVariableCount(basisvalues_size*size());
		AutoDiffVect local_disp(basisvalues_size*size(), 1);

//		DiffScalarBase::setVariableCount(local_dispv.rows());
//		AutoDiffVect local_disp(local_dispv.rows(), 1);
		T energy = T(0.0);

		const AutoDiffAllocator<T> allocate_auto_diff_scalar;

		for (size_t i = 0; i < basisvalues_size*size(); ++i)
		{
			local_disp(i) = allocate_auto_diff_scalar(i, local_dispv[i]);
		}

		AutoDiffGradMat def_grad(size(), size());

		for (long p = 0; p < n_pts; ++p)
		{
			for (long k = 0; k < def_grad.size(); ++k)
				def_grad(k) = T(0);

			for (size_t i = 0; i < vals.basis_values.size(); ++i)
			{
				const auto &bs = vals.basis_values[i];
				const Eigen::Matrix<double, Eigen::Dynamic, 1, 0, 3, 1> grad = bs.grad.row(p);
				assert(grad.size() == size());

				for (int d = 0; d < size(); ++d)
				{
					for (int c = 0; c < size(); ++c)
					{
						def_grad(d, c) += grad(c) * local_disp(i * size() + d);
					}
				}
			}

			AutoDiffGradMat jac_it(size(), size());
			for (long k = 0; k < jac_it.size(); ++k)
				jac_it(k) = T(vals.jac_it[p](k));
			def_grad = def_grad * jac_it;

			//Id + grad d
			for (int d = 0; d < size(); ++d)
				def_grad(d, d) += T(1);

			double lambda, mu;
			params_.lambda_mu(vals.quadrature.points.row(p), vals.val.row(p), vals.element_id, lambda, mu);

			const T log_det_j = log(polyfem::determinant(def_grad));
			const T val = mu / 2 * ((def_grad.transpose() * def_grad).trace() - size() - 2 * log_det_j) + lambda / 2 * log_det_j * log_det_j;

			energy += val * da(p);
		}
		free(local_dispv);
		hipFree(local_dispv_dev);
		return energy;
	}

} // namespace polyfem
