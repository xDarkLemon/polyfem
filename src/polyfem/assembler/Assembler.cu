#include "Assembler.hpp"
#include "CUDA_utilities.cuh"
#include "NeoHookeanElasticity.hpp"
#include "MultiModel.hpp"
#include "hipblas.h"
// #include <polyfem/OgdenElasticity.hpp>

#include <polyfem/utils/Logger.hpp>
//#include <polyfem/MaybeParallelFor.hpp>

#include <igl/Timer.h>

#include <ipc/utils/eigen_ext.hpp>

namespace polyfem
{

	using namespace basis;
	using namespace quadrature;
	using namespace utils;

	namespace assembler
	{

		template <class LocalAssembler>
		double NLAssembler<LocalAssembler>::assemble_GPU(
			const bool is_volume,
			const std::vector<ElementBases> &bases,
			const std::vector<ElementBases> &gbases,
			const AssemblyValsCache &cache,
			const Eigen::MatrixXd &displacement) const
		{
			const int n_bases = int(bases.size());
			double store_val = 0.0;

			//const ElementAssemblyValues* vals_array = cache.access_cache_data();
			std::vector<ElementAssemblyValues> vals_array(n_bases);
			for (int e = 0; e < n_bases; ++e)
			{
				cache.compute(e, is_volume, bases[e], gbases[e], vals_array[e]);
			}
			thrust::device_vector<double> displacement_dev(displacement.col(0).begin(), displacement.col(0).end());

			int jac_it_N = vals_array[0].jac_it.size();
			thrust::device_vector<Eigen::Matrix<double, Eigen::Dynamic, Eigen::Dynamic, 0, 3, 3>> jac_it_dev(n_bases * jac_it_N);

			int basis_values_N = vals_array[0].basis_values.size();
			int global_columns_N = vals_array[0].basis_values[0].global.size();
			thrust::device_vector<basis::Local2Global> global_data_dev(n_bases * basis_values_N * global_columns_N);

			thrust::host_vector<Eigen::Matrix<double, -1, 1, 0, 3, 1>> da_host(n_bases);

			for (int e = 0; e < n_bases; ++e)
			{
				assert(MAX_QUAD_POINTS == -1 || vals_array[e].quadrature.weights.size() < MAX_QUAD_POINTS);
				int N = vals_array[e].det.size();
				da_host[e].resize(N, 1);
				da_host[e] = vals_array[e].det.array() * vals_array[e].quadrature.weights.array();

				thrust::copy(vals_array[e].jac_it.begin(), vals_array[e].jac_it.end(), jac_it_dev.begin() + e * jac_it_N);
				for (int f = 0; f < basis_values_N; f++)
					//needs to be checked
					thrust::copy(vals_array[e].basis_values[f].global.begin(), vals_array[e].basis_values[f].global.end(), global_data_dev.begin() + e * (basis_values_N * global_columns_N) + f * global_columns_N);
			}

			thrust::device_vector<Eigen::Matrix<double, -1, 1, 0, 3, 1>> da_dev(n_bases);
			thrust::copy(da_host.begin(), da_host.end(), da_dev.begin());

			double lambda, mu;
			const int n_pts = da_host[0].size();

			thrust::device_vector<Eigen::Matrix<double, -1, 1, 0, 3, 1>> grad_dev(n_bases * basis_values_N * n_pts);
			for (int e = 0; e < n_bases; ++e)
			{
				for (int f = 0; f < basis_values_N; f++)
				{
					for (int p = 0; p < n_pts; p++)
						grad_dev[e * basis_values_N * n_pts + f * n_pts + p] = vals_array[e].basis_values[f].grad.row(p);
					//					thrust::copy(vals_array[e].basis_values[f].grad.row(p).begin(),vals_array[e].basis_values[f].grad.row(p).end(), grad_dev.begin()+e*(basis_values_N*global_columns_N)+f*global_columns_N+p);
				}
			}

			// extract all lambdas and mus and set to device vector
			thrust::device_vector<double> lambda_array(n_pts);
			thrust::device_vector<double> mu_array(n_pts);
			for (int p = 0; p < n_pts; p++)
			{
				local_assembler_.get_lambda_mu(vals_array[0].quadrature.points.row(p), vals_array[0].val.row(p), vals_array[0].element_id, lambda, mu);
				lambda_array[p] = lambda;
				mu_array[p] = mu;
			}

			// READY TO SEND ALL TO GPU

			double *displacement_dev_ptr = thrust::raw_pointer_cast(displacement_dev.data());
			Eigen::Matrix<double, Eigen::Dynamic, Eigen::Dynamic, 0, 3, 3> *jac_it_dev_ptr = thrust::raw_pointer_cast(jac_it_dev.data());
			basis::Local2Global *global_data_dev_ptr = thrust::raw_pointer_cast(global_data_dev.data());
			Eigen::Matrix<double, -1, 1, 0, 3, 1> *da_dev_ptr = thrust::raw_pointer_cast(da_dev.data());
			Eigen::Matrix<double, -1, 1, 0, 3, 1> *grad_dev_ptr = thrust::raw_pointer_cast(grad_dev.data());

			double *lambda_ptr = thrust::raw_pointer_cast(lambda_array.data());
			double *mu_ptr = thrust::raw_pointer_cast(mu_array.data());

			thrust::device_vector<double> energy_dev_storage(n_bases, double(0.0));
			double *energy_dev_storage_ptr = thrust::raw_pointer_cast(energy_dev_storage.data());

			hipDeviceSynchronize();
			local_assembler_.compute_energy_gpu(displacement_dev_ptr,
												jac_it_dev_ptr,
												global_data_dev_ptr,
												da_dev_ptr,
												grad_dev_ptr,
												n_bases,
												basis_values_N,
												global_columns_N,
												n_pts,
												lambda_ptr,
												mu_ptr,
												energy_dev_storage_ptr);

			hipDeviceSynchronize();
			thrust::host_vector<double> energy_stg(energy_dev_storage.begin(), energy_dev_storage.end());
			double init = 0.0;

			store_val = thrust::reduce(energy_stg.begin(), energy_stg.end(), init, thrust::plus<double>());

			return store_val;
		}

		template <class LocalAssembler>
		void NLAssembler<LocalAssembler>::assemble_grad_GPU(
			const bool is_volume,
			const int n_basis,
			const std::vector<ElementBases> &bases,
			const std::vector<ElementBases> &gbases,
			const AssemblyValsCache &cache,
			const Eigen::MatrixXd &displacement,
			Eigen::MatrixXd &rhs) const
		{
			rhs.resize(n_basis * local_assembler_.size(), 1);
			rhs.setZero();

			const int n_bases = int(bases.size());
			Eigen::MatrixXd vec;
			vec.resize(rhs.size(), 1);
			vec.setZero();

			std::vector<ElementAssemblyValues> vals_array(n_bases);

			for (int e = 0; e < n_bases; ++e)
			{
				cache.compute(e, is_volume, bases[e], gbases[e], vals_array[e]);
			}
			thrust::device_vector<double> displacement_dev(displacement.col(0).begin(), displacement.col(0).end());
			int jac_it_N = vals_array[0].jac_it.size();

			thrust::device_vector<Eigen::Matrix<double, Eigen::Dynamic, Eigen::Dynamic, 0, 3, 3>> jac_it_dev(n_bases * jac_it_N);

			int basis_values_N = vals_array[0].basis_values.size();
			int global_columns_N = vals_array[0].basis_values[0].global.size();
			thrust::device_vector<basis::Local2Global> global_data_dev(n_bases * basis_values_N * global_columns_N);

			thrust::host_vector<Eigen::Matrix<double, -1, 1, 0, 3, 1>> da_host(n_bases);

			for (int e = 0; e < n_bases; ++e)
			{
				assert(MAX_QUAD_POINTS == -1 || quadrature.weights.size() < MAX_QUAD_POINTS);
				int N = vals_array[e].det.size();
				da_host[e].resize(N, 1);
				da_host[e] = vals_array[e].det.array() * vals_array[e].quadrature.weights.array();

				thrust::copy(vals_array[e].jac_it.begin(), vals_array[e].jac_it.end(), jac_it_dev.begin() + e * jac_it_N);
				for (int f = 0; f < basis_values_N; f++)
				{
					//needs a paranoic check
					thrust::copy(vals_array[e].basis_values[f].global.begin(), vals_array[e].basis_values[f].global.end(), global_data_dev.begin() + e * (basis_values_N * global_columns_N) + f * global_columns_N);
				}
			}

			thrust::device_vector<Eigen::Matrix<double, -1, 1, 0, 3, 1>> da_dev(n_bases);
			thrust::copy(da_host.begin(), da_host.end(), da_dev.begin());

			const int n_pts = da_host[0].size();

			thrust::device_vector<Eigen::Matrix<double, -1, -1, 0, 3, 3>> grad_dev(n_bases * basis_values_N * n_pts);
			for (int e = 0; e < n_bases; ++e)
			{
				for (int f = 0; f < basis_values_N; f++)
				{
					for (int p = 0; p < n_pts; p++)
						grad_dev[e * basis_values_N * n_pts + f * n_pts + p] = vals_array[e].basis_values[f].grad.row(p);
				}
			}

			// extract all lambdas and mus and set to device vector
			double lambda, mu;
			thrust::device_vector<double> lambda_array(n_pts);
			thrust::device_vector<double> mu_array(n_pts);
			for (int p = 0; p < n_pts; p++)
			{
				local_assembler_.get_lambda_mu(vals_array[0].quadrature.points.row(p), vals_array[0].val.row(p), vals_array[0].element_id, lambda, mu);
				lambda_array[p] = lambda;
				mu_array[p] = mu;
			}

			// READY TO SEND ALL TO GPU

			double *displacement_dev_ptr = thrust::raw_pointer_cast(displacement_dev.data());

			Eigen::Matrix<double, Eigen::Dynamic, Eigen::Dynamic, 0, 3, 3> *jac_it_dev_ptr = thrust::raw_pointer_cast(jac_it_dev.data());
			basis::Local2Global *global_data_dev_ptr = thrust::raw_pointer_cast(global_data_dev.data());
			Eigen::Matrix<double, -1, 1, 0, 3, 1> *da_dev_ptr = thrust::raw_pointer_cast(da_dev.data());
			Eigen::Matrix<double, -1, -1, 0, 3, 3> *grad_dev_ptr = thrust::raw_pointer_cast(grad_dev.data());

			double *lambda_ptr = thrust::raw_pointer_cast(lambda_array.data());
			double *mu_ptr = thrust::raw_pointer_cast(mu_array.data());

			vec = local_assembler_.assemble_grad_GPU(displacement_dev_ptr,
													 jac_it_dev_ptr,
													 global_data_dev_ptr,
													 da_dev_ptr,
													 grad_dev_ptr,
													 n_bases,
													 basis_values_N,
													 global_columns_N,
													 n_pts,
													 lambda_ptr,
													 mu_ptr,
													 n_basis);
			rhs += vec;
		}

		//template instantiation
		template class NLAssembler<NeoHookeanElasticity>;
	} // namespace assembler
} // namespace polyfem
