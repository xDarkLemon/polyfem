#include "hip/hip_runtime.h"
#pragma once

#include <polyfem/Common.hpp>
#include <polyfem/utils/Logger.hpp>
#include <polyfem/utils/Timer.hpp>
#include <polyfem/Common.hpp>

#include <polyfem/solver/NLProblem.hpp>
#include <polyfem/solver/FullNLProblem.hpp>
#include <polyfem/solver/SparseNewtonDescentSolver.hpp>
#include "polyfem/utils/CUDA_utilities.cuh"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include "hipsparse.h"
#include "hip/library_types.h"

namespace cppoptlib
{
    void EigenSparseToCuSparseTranspose(const Eigen::SparseMatrix<double> &mat, int *row, int *col, double *val)
    {
        const int num_non0  = mat.nonZeros();
        const int num_outer = mat.cols() + 1;
        hipMemcpy(row, mat.outerIndexPtr(), sizeof(int) * num_outer, hipMemcpyHostToDevice);
        hipMemcpy(col, mat.innerIndexPtr(), sizeof(int) * num_non0, hipMemcpyHostToDevice);
        hipMemcpy(val, mat.valuePtr(), sizeof(double) * num_non0, hipMemcpyHostToDevice);
    }

	template <typename ProblemType>
	bool SparseNewtonDescentSolver<ProblemType>::check_direction_gpu(
		const polyfem::StiffnessMatrix &hessian,
		const Eigen::Matrix<double, -1, 1> &grad,
		const Eigen::Matrix<double, -1, 1> &direction)
	{
        int N = hessian.cols();
        
        double *hessian_dev, *direction_dev, *grad_dev, *tmp_dev, *residual_dev;  // to compute residual
        double *grad_norm_dev, *grad_dir_dot_dev;  // to compute grad norm and grad dot direction
        
        double *residual_h = new double[1];
        double *grad_dir_dot_h = new double[1];
        double *grad_norm_h = new double[1];
        
        // move direction, grad to gpu
        direction_dev = ALLOCATE_GPU<double>(direction_dev, N*sizeof(double));
        grad_dev = ALLOCATE_GPU<double>(grad_dev, N*sizeof(double));
        COPYDATATOGPU<double>(direction_dev, direction.data(), N*sizeof(double));
        COPYDATATOGPU<double>(grad_dev, grad.data(), N*sizeof(double));

        // move hessian to gpu (compressed format)
        const int non0 = hessian.nonZeros();
        polyfem::logger().trace("non0: {}, cols: {}, rows: {}, allocating size: {} bytes", non0,  hessian.cols(), hessian.rows(), non0*sizeof(double));
        int *row_dev, *col_dev;
        row_dev = ALLOCATE_GPU<int>(row_dev, (N+1)*sizeof(int));
        col_dev = ALLOCATE_GPU<int>(col_dev, non0*sizeof(int));
        hessian_dev = ALLOCATE_GPU<double>(hessian_dev, non0*sizeof(double));
        EigenSparseToCuSparseTranspose(hessian, row_dev, col_dev, hessian_dev);
        
        // compute residual
        // const double residual = (hessian * direction + grad).norm(); // H Δx + g = 0
        tmp_dev = ALLOCATE_GPU<double>(tmp_dev, N*sizeof(double));
        COPYDATATOGPU<double>(tmp_dev, grad.data(), N*sizeof(double));
        residual_dev = ALLOCATE_GPU<double>(residual_dev, sizeof(double));

        hipsparseStatus_t status;
        hipsparseHandle_t handle=0;
        hipsparseMatDescr_t descr=0;
        status= hipsparseCreate(&handle);
        status= hipsparseCreateMatDescr(&descr);
        hipsparseSetMatType(descr , HIPSPARSE_MATRIX_TYPE_GENERAL);
        hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO); 
        double *buffer;
        buffer = ALLOCATE_GPU<double>(buffer, 2*non0*sizeof(double));
        double alpha = 1.0;
        double beta = 1.0;
        status=cusparseCsrmvEx(handle, CUSPARSE_ALG_MERGE_PATH, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, N, non0, &alpha , HIP_R_64F, descr , hessian_dev, HIP_R_64F, row_dev , col_dev , direction_dev , HIP_R_64F, &beta , HIP_R_64F, tmp_dev, HIP_R_64F, HIP_R_64F, buffer);
        hipsparseDestroyMatDescr(descr);
        hipsparseDestroy(handle); 
        hipFree(row_dev);
        hipFree(col_dev);
        hipFree(buffer);

        hipblasHandle_t handle2;
        hipblasCreate(&handle2);
        hipblasDnrm2(handle2, N, tmp_dev, 1, residual_h);

        // compute grad norm, grad direction dot product
        grad_norm_dev = ALLOCATE_GPU<double>(grad_norm_dev, sizeof(double));
        grad_dir_dot_dev = ALLOCATE_GPU<double>(grad_dir_dot_dev, sizeof(double));
        
        hipblasDnrm2(handle2, N, grad_dev, 1, grad_norm_h);
        hipblasDdot(handle2, N, grad_dev, 1, direction_dev, 1, grad_dir_dot_h);
        hipblasDestroy(handle2);
        
        hipFree(hessian_dev);
        hipFree(direction_dev);
        hipFree(grad_dev);
        hipFree(tmp_dev);
        hipFree(residual_dev);
        hipFree(grad_norm_dev);
        hipFree(grad_dir_dot_dev);

        const double residual = *residual_h;
        const double grad_norm = *grad_norm_h;
        const double grad_dir_dot = *grad_dir_dot_h;

        delete[] residual_h;
        delete[] grad_norm_h;
        delete[] grad_dir_dot_h;

        // gradient descent, check descent direction
        if (std::isnan(residual))
        {
            increase_descent_strategy();
            polyfem::logger().log(
                this->descent_strategy == 2 ? spdlog::level::warn : spdlog::level::debug,
                "nan linear solve residual {} (||∇f||={}); reverting to {}",
                residual, grad_norm, this->descent_strategy_name());
            return false;
        }
        else if (residual > std::max(1e-8 * grad_norm, 1e-5))
        {
            increase_descent_strategy();
            polyfem::logger().log(
                this->descent_strategy == 2 ? spdlog::level::warn : spdlog::level::debug,
                "large linear solve residual {} (||∇f||={}); reverting to {}",
                residual, grad_norm, this->descent_strategy_name());
            return false;
        }
        else
        {
            polyfem::logger().trace("linear solve residual {}", residual);
        }

        // do this check here because we need to repeat the solve without resetting reg_weight
        if (grad_dir_dot >= 0)
        {
            increase_descent_strategy();
            polyfem::logger().log(
                this->descent_strategy == 2 ? spdlog::level::warn : spdlog::level::debug,
                "[{}] direction is not a descent direction (Δx⋅g={}≥0); reverting to {}",
                name(), grad_dir_dot, descent_strategy_name());
            return false;
        }

        return true;
    }
    template class SparseNewtonDescentSolver<polyfem::solver::NLProblem>;
    template class SparseNewtonDescentSolver<polyfem::solver::FullNLProblem>;
}
